#include "hip/hip_runtime.h"
/*
  GridTools Libraries

  Copyright (c) 2017, ETH Zurich and MeteoSwiss
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

  1. Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

  3. Neither the name of the copyright holder nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  For information: http://eth-cscs.github.io/gridtools/
*/
#include "gtest/gtest.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "common/gpu_clone.hpp"
#include "storage/hybrid_pointer.hpp"

using gridtools::uint_t;
using gridtools::int_t;

struct A : gridtools::clonable_to_gpu< A > {
    gridtools::hybrid_pointer< int > p;

    A(uint_t n) : p(n) {
#ifndef NDEBUG
        p.out();
#endif
    }

    __device__ A(A const &other) : p(other.p) {
#ifndef NDEBUG
        p.out();
#endif
    }
};

__global__ void reverse(A *p, uint_t n) {
#ifndef NDEBUG
    if (p->p.on_host())
        printf(" cpu_p %X ", p->p.get_cpu_p());
    if (p->p.on_device())
        printf(" gpu_p %X ", p->p.get_gpu_p());
    printf(" to_use %X ", p->p.get_pointer_to_use());
    printf(" siez %X ", p->p.get_size());
    printf("\n");
#endif
    for (uint_t i = 0; i < p->p.get_size(); ++i)
        p->p[i] = n - i;
}

bool test_hybrid_pointer() {
    uint_t n = 10;
    A a(n);

    for (uint_t i = 0; i < n; ++i)
        a.p[i] = i;

    a.p.update_gpu();
    a.clone_to_device();

    // clang-format off
    reverse<<<1,1>>>(a.gpu_object_ptr, n);
    // clang-format on

    hipDeviceSynchronize();

    a.p.update_cpu();

    bool right = true;
    for (uint_t i = 0; i < n; ++i)
        if (a.p[i] != n - i)
            right = false;

    return right;
}

TEST(test_hybrid_pointer, hybrid_pointer_on_gpu) { EXPECT_EQ(test_hybrid_pointer(), true); }
