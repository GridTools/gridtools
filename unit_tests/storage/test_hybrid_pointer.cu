#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "common/gpu_clone.hpp"
#include "storage/hybrid_pointer.hpp"

using gridtools::uint_t;
using gridtools::int_t;

struct A : gridtools::clonable_to_gpu< A > {
    gridtools::hybrid_pointer< int > p;

    A(uint_t n) : p(n, false) {
#ifndef NDEBUG
        p.out();
#endif
    }

    __device__ A(A const &other) : p(other.p) {
#ifndef NDEBUG
        p.out();
#endif
    }
};

__global__ void reverse(A *p, uint_t n) {
#ifndef NDEBUG
    if (p->p.on_host())
        printf(" cpu_p %X ", p->p.get_cpu_p());
    if (p->p.on_device())
        printf(" gpu_p %X ", p->p.get_gpu_p());
    printf(" to_use %X ", p->p.get_pointer_to_use());
    printf(" siez %X ", p->p.get_size());
    printf("\n");
#endif
    for (uint_t i = 0; i < p->p.get_size(); ++i)
        p->p[i] = n - i;
}

bool test_hybrid_pointer() {
    uint_t n = 10;
    A a(n);

    for (uint_t i = 0; i < n; ++i)
        a.p[i] = i;

    a.p.update_gpu();
    a.clone_to_device();

    // clang-format off
    reverse<<<1,1>>>(a.gpu_object_ptr, n);
    // clang-format on

    hipDeviceSynchronize();

    a.p.update_cpu();

    bool right = true;
    for (uint_t i = 0; i < n; ++i)
        if (a.p[i] != n - i)
            right = false;

    return right;
}

TEST(test_hybrid_pointer, hybrid_pointer_on_gpu) { EXPECT_EQ(test_hybrid_pointer(), true); }
