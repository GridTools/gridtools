#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include "gtest/gtest.h"

#include <gridtools/common/gt_assert.hpp>
#include <gridtools/common/variadic_pack_metafunctions.hpp>
#include <gridtools/storage/data_store.hpp>
#include <gridtools/storage/storage_cuda/cuda_storage.hpp>
#include <gridtools/storage/storage_cuda/cuda_storage_info.hpp>

using namespace gridtools;

typedef storage_info<0, layout_map<2, 1, 0>, halo<0, 0, 0>, alignment<32>> storage_info_t;

__global__ void mul2(double *s) {
    s[0] *= 2.0;
    s[1] *= 2.0;
}

template <typename StorageInfo>
__global__ void check_vals(double *s, StorageInfo const *si) {
    for (uint_t i = 0; i < si->template total_length<0>(); ++i)
        for (uint_t j = 0; j < si->template total_length<1>(); ++j)
            for (uint_t k = 0; k < si->template total_length<2>(); ++k) {
                int x = si->index(i, j, k);
                if (s[x] > 3.141499 && s[x] < 3.141501) {
                    s[x] = 1.0;
                } else {
                    s[x] = 0.0;
                }
            }
}

template <typename StorageInfo>
__global__ void check_vals_lambda(double *s, StorageInfo const *si) {
    for (uint_t i = 0; i < si->template total_length<0>(); ++i)
        for (uint_t j = 0; j < si->template total_length<1>(); ++j)
            for (uint_t k = 0; k < si->template total_length<2>(); ++k) {
                int x = si->index(i, j, k);
                if (s[x] == i + j + k) {
                    s[x] = i + j + k + 1.0;
                } else {
                    s[x] = 0.0;
                }
            }
}

TEST(DataStoreTest, Simple) {
    using data_store_t = data_store<cuda_storage<double>, storage_info_t>;
    storage_info_t si(3, 3, 3);

    constexpr storage_info<0, layout_map<2, 1, 0>> csi(3, 3, 3);
    constexpr storage_info<1, layout_map<2, 1, 0>, halo<2, 1, 0>> csih(7, 5, 3);
    constexpr storage_info<2, layout_map<2, 1, 0>, halo<2, 1, 0>, alignment<16>> csiha(7, 5, 3);

    // check sizes, strides, and alignment
    EXPECT_EQ(csi.total_length<0>(), 3);
    EXPECT_EQ(csi.total_length<1>(), 3);
    EXPECT_EQ(csi.total_length<2>(), 3);

    EXPECT_EQ(csi.stride<0>(), 1);
    EXPECT_EQ(csi.stride<1>(), 3);
    EXPECT_EQ(csi.stride<2>(), 9);

    EXPECT_EQ(csih.total_length<0>(), 7);
    EXPECT_EQ(csih.total_length<1>(), 5);
    EXPECT_EQ(csih.total_length<2>(), 3);

    EXPECT_EQ(csih.stride<0>(), 1);
    EXPECT_EQ(csih.stride<1>(), 7);
    EXPECT_EQ(csih.stride<2>(), 35);

    EXPECT_EQ(csiha.total_length<0>(), 7);
    EXPECT_EQ(csiha.total_length<1>(), 5);
    EXPECT_EQ(csiha.total_length<2>(), 3);

    EXPECT_EQ(csiha.padded_length<0>(), 16);
    EXPECT_EQ(csiha.padded_length<1>(), 5);
    EXPECT_EQ(csiha.padded_length<2>(), 3);

    EXPECT_EQ(csiha.stride<0>(), 1);
    EXPECT_EQ(csiha.stride<1>(), 16);
    EXPECT_EQ(csiha.stride<2>(), 16 * 5);

    data_store_t ds(si);
    data_store_t ds_tmp_1(si);
    data_store_t ds_tmp_2 = ds; // copy construct
    ds_tmp_1 = ds;              // copy assign

    // create a copy of a data_store and check equivalence
    data_store_t datast(si);
    data_store_t datast_cpy(datast);
    EXPECT_EQ(&datast.info(), &datast_cpy.info());
    EXPECT_EQ(&datast.storage(), &datast_cpy.storage());

    // modify the data and check if the copy can see the changes
    datast.storage().get_cpu_ptr()[0] = 100;
    datast.storage().get_cpu_ptr()[1] = 200;
    EXPECT_EQ(datast.storage().get_cpu_ptr()[0], 100);
    EXPECT_EQ(datast.storage().get_cpu_ptr()[1], 200);
    EXPECT_EQ(datast_cpy.storage().get_cpu_ptr()[0], 100);
    EXPECT_EQ(datast_cpy.storage().get_cpu_ptr()[1], 200);

    // clone to device
    datast.clone_to_device();
    mul2<<<1, 1>>>(datast.storage().get_target_ptr());

    // check again
    datast.storage().get_cpu_ptr()[0] = 200;
    datast.storage().get_cpu_ptr()[1] = 400;
    EXPECT_EQ(datast.storage().get_cpu_ptr()[0], 200);
    EXPECT_EQ(datast.storage().get_cpu_ptr()[1], 400);
    EXPECT_EQ(datast_cpy.storage().get_cpu_ptr()[0], 200);
    EXPECT_EQ(datast_cpy.storage().get_cpu_ptr()[1], 400);

    data_store<cuda_storage<double>, storage_info_t> ds_cpy_ass1(si);
    data_store<cuda_storage<double>, storage_info_t> ds_cpy_ass2 = ds_cpy_ass1;
    EXPECT_TRUE(ds_cpy_ass2.storage().get_cpu_ptr() == ds_cpy_ass1.storage().get_cpu_ptr());
    EXPECT_TRUE(ds_cpy_ass2.storage().get_target_ptr() == ds_cpy_ass1.storage().get_target_ptr());
    EXPECT_TRUE(ds_cpy_ass2.info() == ds_cpy_ass1.info());
}

TEST(DataStoreTest, States) {
    using data_store_t = data_store<cuda_storage<double>, storage_info_t>;
    storage_info_t si(3, 3, 3);
    // create and allocate data_store
    data_store_t ds(si);

    // intial state should be
    EXPECT_FALSE(ds.storage().state().host_needs_update());
    EXPECT_FALSE(ds.storage().state().device_needs_update());

    // host write views should be valid, this means the device needs an update
    ds.reactivate_host_write_views();
    EXPECT_FALSE(ds.storage().state().host_needs_update());
    EXPECT_TRUE(ds.storage().state().device_needs_update());

    // synchronize everything, valid on both sides
    ds.sync();
    EXPECT_FALSE(ds.storage().state().host_needs_update());
    EXPECT_FALSE(ds.storage().state().device_needs_update());

    // device write views should be valid, this means the host needs an update
    ds.reactivate_target_write_views();
    EXPECT_TRUE(ds.storage().state().host_needs_update());
    EXPECT_FALSE(ds.storage().state().device_needs_update());

    // go back from device
    ds.sync();
    EXPECT_FALSE(ds.storage().state().host_needs_update());
    EXPECT_FALSE(ds.storage().state().device_needs_update());
}

TEST(DataStoreTest, Initializer) {
    storage_info_t si(12, 12, 8);
    data_store<cuda_storage<double>, storage_info_t> ds(si, 3.1415);
    check_vals<<<1, 1>>>(ds.storage().get_target_ptr(), get_gpu_storage_info_ptr(ds.info()));
    ds.clone_from_device();
    for (uint_t i = 0; i < 12; ++i)
        for (uint_t j = 0; j < 12; ++j)
            for (uint_t k = 0; k < 8; ++k)
                EXPECT_EQ(ds.storage().get_cpu_ptr()[si.index(i, j, k)], 1.0);
}

TEST(DataStoreTest, LambdaInitializer) {
    storage_info_t si(10, 11, 12);
    data_store<cuda_storage<double>, storage_info_t> ds(si, [](int i, int j, int k) { return i + j + k; });
    check_vals_lambda<<<1, 1>>>(ds.storage().get_target_ptr(), get_gpu_storage_info_ptr(ds.info()));
    ds.clone_from_device();
    for (uint_t i = 0; i < 10; ++i)
        for (uint_t j = 0; j < 11; ++j)
            for (uint_t k = 0; k < 12; ++k)
                EXPECT_EQ(ds.storage().get_cpu_ptr()[si.index(i, j, k)], (i + j + k + 1.0));
}

TEST(DataStoreTest, Naming) {
    storage_info_t si(10, 11, 12);
    // no naming
    data_store<cuda_storage<double>, storage_info_t> ds2_nn(si);
    data_store<cuda_storage<double>, storage_info_t> ds3_nn(si, 1.0);
    data_store<cuda_storage<double>, storage_info_t> ds4_nn(si, [](int i, int j, int k) { return i + j + k; });
    EXPECT_EQ(ds2_nn.name(), "");
    EXPECT_EQ(ds3_nn.name(), "");
    EXPECT_EQ(ds4_nn.name(), "");

    // test naming
    data_store<cuda_storage<double>, storage_info_t> ds2(si, "standard storage");
    data_store<cuda_storage<double>, storage_info_t> ds3(si, 1.0, "value init. storage");
    data_store<cuda_storage<double>, storage_info_t> ds4(
        si, [](int i, int j, int k) { return i + j + k; }, "lambda init. storage");
    EXPECT_EQ(ds2.name(), "standard storage");
    EXPECT_EQ(ds3.name(), "value init. storage");
    EXPECT_EQ(ds4.name(), "lambda init. storage");

    // create a copy and see if still ok
    auto ds2_tmp = ds2;
    EXPECT_EQ(ds2_tmp.name(), "standard storage");
    EXPECT_EQ(ds3.name(), "value init. storage");
}

TEST(DataStoreTest, ExternalPointer) {
    // test with an external CPU pointer
    storage_info_t si(10, 10, 10);
    double *external_ptr = new double[si.padded_total_length()];
    // create a data_store with externally managed storage
    data_store<cuda_storage<double>, storage_info_t> ds(si, external_ptr, ownership::external_cpu);
    ds.sync();
    // create a copy (double free checks)
    data_store<cuda_storage<double>, storage_info_t> ds_cpy = ds;
    // check values
    for (uint_t i = 0; i < 10; ++i)
        for (uint_t j = 0; j < 10; ++j)
            for (uint_t k = 0; k < 10; ++k) {
                external_ptr[si.index(i, j, k)] = 3.1415;
                EXPECT_EQ(ds.storage().get_cpu_ptr()[si.index(i, j, k)], 3.1415);
                EXPECT_EQ(ds_cpy.storage().get_cpu_ptr()[si.index(i, j, k)], 3.1415);
            }
    // delete the ptr
    delete[] external_ptr;
}

TEST(DataStoreTest, DimAndSizeInterface) {
    storage_info_t si(128, 128, 80);
    data_store<cuda_storage<double>, storage_info_t> ds(si, 3.1415);
    EXPECT_EQ(ds.padded_total_length(), si.padded_total_length());
    EXPECT_EQ(ds.total_length<0>(), si.total_length<0>());
    EXPECT_EQ(ds.total_length<1>(), si.total_length<1>());
    EXPECT_EQ(ds.total_length<2>(), si.total_length<2>());
}

TEST(DataStoreTest, ExternalGPUPointer) {
    // test with an external GPU pointer
    storage_info_t si(10, 10, 10);
    double *external_gpu_ptr;
    double *external_cpu_ptr = new double[si.padded_total_length()];
    // initialize CPU ptr
    for (uint_t i = 0; i < si.padded_total_length(); ++i) {
        external_cpu_ptr[i] = 3.1415;
    }
    // create a GPU ptr
    GT_CUDA_CHECK(hipMalloc(&external_gpu_ptr, si.padded_total_length() * sizeof(double)));
    // initialize the GPU ptr
    GT_CUDA_CHECK(hipMemcpy(
        external_gpu_ptr, external_cpu_ptr, si.padded_total_length() * sizeof(double), hipMemcpyHostToDevice));
    // create a data_store with externally managed storage
    data_store<cuda_storage<double>, storage_info_t> ds(si, external_gpu_ptr, ownership::external_gpu);
    ds.sync();
    // create some copies
    data_store<cuda_storage<double>, storage_info_t> ds_cpy_1(ds);
    data_store<cuda_storage<double>, storage_info_t> ds_cpy_2 = ds_cpy_1;
    EXPECT_EQ(ds_cpy_1.storage().get_cpu_ptr(), ds_cpy_2.storage().get_cpu_ptr());
    EXPECT_EQ(ds_cpy_2.storage().get_cpu_ptr(), ds.storage().get_cpu_ptr());
    // create a copy (double free checks)
    data_store<cuda_storage<double>, storage_info_t> ds_cpy = ds;
    // check values
    for (uint_t i = 0; i < 10; ++i)
        for (uint_t j = 0; j < 10; ++j)
            for (uint_t k = 0; k < 10; ++k) {
                EXPECT_EQ(ds.storage().get_cpu_ptr()[si.index(i, j, k)], 3.1415);
                EXPECT_EQ(ds_cpy.storage().get_cpu_ptr()[si.index(i, j, k)], 3.1415);
            }
    // delete the ptr
    delete[] external_cpu_ptr;
    GT_CUDA_CHECK(hipFree(external_gpu_ptr));
}
