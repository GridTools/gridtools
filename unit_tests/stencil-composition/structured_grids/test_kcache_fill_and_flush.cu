/*
  GridTools Libraries

  Copyright (c) 2017, ETH Zurich and MeteoSwiss
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

  1. Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

  3. Neither the name of the copyright holder nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  For information: http://eth-cscs.github.io/gridtools/
*/
#include "kcache_fixture.hpp"
#include "gtest/gtest.h"
#include <gridtools/stencil-composition/stencil-composition.hpp>
#include <gridtools/tools/verifier.hpp>

using namespace gridtools;
using namespace enumtype;
using namespace expressions;

// These are the stencil operators that compose the multistage stencil in this test
struct shift_acc_forward_fill_and_flush {

    typedef accessor<0, enumtype::inout, extent<0, 0, 0, 0, -1, 0>> in;

    typedef boost::mpl::vector<in> arg_list;

    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kbody_high) {
        eval(in()) = eval(in()) + eval(in(0, 0, -1));
    }
    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kminimum) {
        eval(in()) = eval(in());
    }
};

struct shift_acc_backward_fill_and_flush {

    typedef accessor<0, enumtype::inout, extent<0, 0, 0, 0, 0, 1>> in;

    typedef boost::mpl::vector<in> arg_list;

    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kbody_low) {
        eval(in()) = eval(in()) + eval(in(0, 0, 1));
    }
    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kmaximum) {
        eval(in()) = eval(in());
    }
};

struct copy_fill {

    typedef accessor<0, enumtype::inout> in;

    typedef boost::mpl::vector<in> arg_list;

    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kfull) {
        eval(in()) = eval(in());
    }
};

struct scale_fill {

    typedef accessor<0, enumtype::inout> in;

    typedef boost::mpl::vector<in> arg_list;

    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kfull) {
        eval(in()) = 2 * eval(in());
    }
};

TEST_F(kcachef, fill_and_flush_forward) {

    for (uint_t i = 0; i < m_d1; ++i) {
        for (uint_t j = 0; j < m_d2; ++j) {
            m_refv(i, j, 0) = m_inv(i, j, 0);
            for (uint_t k = 1; k < m_d3; ++k) {
                m_refv(i, j, k) = m_inv(i, j, k) + m_refv(i, j, k - 1);
            }
        }
    }

    typedef arg<0, storage_t> p_in;

    auto kcache_stencil = gridtools::make_computation<backend_t>(m_grid,
        p_in{} = m_in,
        gridtools::make_multistage // mss_descriptor
        (execute<forward>(),
            define_caches(cache<K, cache_io_policy::fill_and_flush, kfull>(p_in())),
            gridtools::make_stage<shift_acc_forward_fill_and_flush>(p_in())));

    kcache_stencil.run();

#if FLOAT_PRECISION == 4
    verifier verif(1e-6);
#else
    verifier verif(1e-10);
#endif
    array<array<uint_t, 2>, 3> halos{{{0, 0}, {0, 0}, {0, 0}}};

    m_in.sync();
    ASSERT_TRUE(verif.verify(m_grid, m_ref, m_in, halos));
}

TEST_F(kcachef, fill_and_flush_backward) {

    for (uint_t i = 0; i < m_d1; ++i) {
        for (uint_t j = 0; j < m_d2; ++j) {
            m_refv(i, j, m_d3 - 1) = m_inv(i, j, m_d3 - 1);
            for (int_t k = m_d3 - 2; k >= 0; --k) {
                m_refv(i, j, k) = m_refv(i, j, k + 1) + m_inv(i, j, k);
            }
        }
    }

    typedef arg<0, storage_t> p_in;

    auto kcache_stencil = gridtools::make_computation<backend_t>(m_grid,
        p_in{} = m_in,
        gridtools::make_multistage(execute<backward>(),
            define_caches(cache<K, cache_io_policy::fill_and_flush, kfull>(p_in())),
            gridtools::make_stage<shift_acc_backward_fill_and_flush>(p_in())));

    kcache_stencil.run();

#if FLOAT_PRECISION == 4
    verifier verif(1e-6);
#else
    verifier verif(1e-10);
#endif
    array<array<uint_t, 2>, 3> halos{{{0, 0}, {0, 0}, {0, 0}}};

    m_in.sync();
    ASSERT_TRUE(verif.verify(m_grid, m_ref, m_in, halos));
}

TEST_F(kcachef, fill_copy_forward) {

    for (uint_t i = 0; i < m_d1; ++i) {
        for (uint_t j = 0; j < m_d2; ++j) {
            for (uint_t k = 0; k < m_d3; ++k) {
                m_refv(i, j, k) = m_inv(i, j, k);
            }
        }
    }

    typedef arg<0, storage_t> p_in;

    auto kcache_stencil = gridtools::make_computation<backend_t>(m_grid,
        p_in{} = m_in,
        gridtools::make_multistage(execute<forward>(),
            define_caches(cache<K, cache_io_policy::fill_and_flush, kfull>(p_in())),
            gridtools::make_stage<copy_fill>(p_in())));

    kcache_stencil.run();

#if FLOAT_PRECISION == 4
    verifier verif(1e-6);
#else
    verifier verif(1e-10);
#endif
    array<array<uint_t, 2>, 3> halos{{{0, 0}, {0, 0}, {0, 0}}};

    m_in.sync();
    ASSERT_TRUE(verif.verify(m_grid, m_ref, m_in, halos));
}

TEST_F(kcachef, fill_scale_forward) {

    for (uint_t i = 0; i < m_d1; ++i) {
        for (uint_t j = 0; j < m_d2; ++j) {
            for (uint_t k = 0; k < m_d3; ++k) {
                m_refv(i, j, k) = 2 * m_inv(i, j, k);
            }
        }
    }

    typedef arg<0, storage_t> p_in;

    auto kcache_stencil = gridtools::make_computation<backend_t>(m_grid,
        p_in{} = m_in,
        gridtools::make_multistage(execute<forward>(),
            define_caches(cache<K, cache_io_policy::fill_and_flush, kfull>(p_in())),
            gridtools::make_stage<scale_fill>(p_in())));

    kcache_stencil.run();

#if FLOAT_PRECISION == 4
    verifier verif(1e-6);
#else
    verifier verif(1e-10);
#endif
    array<array<uint_t, 2>, 3> halos{{{0, 0}, {0, 0}, {0, 0}}};

    m_in.sync();
    ASSERT_TRUE(verif.verify(m_grid, m_ref, m_in, halos));
}

struct do_nothing {

    typedef accessor<0, enumtype::inout, extent<0, 0, 0, 0, -1, 1>> in;

    typedef boost::mpl::vector<in> arg_list;

    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kminimum) {}
    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kmaximum) {}
    template <typename Evaluation>
    GT_FUNCTION static void Do(Evaluation &eval, kbody) {}
};

TEST_F(kcachef, fill_copy_forward_with_extent) {

    for (uint_t i = 0; i < m_d1; ++i) {
        for (uint_t j = 0; j < m_d2; ++j) {
            for (uint_t k = 0; k < m_d3; ++k) {
                m_refv(i, j, k) = m_inv(i, j, k) = k;
            }
        }
    }
    m_in.sync();
    m_ref.sync();

    typedef arg<0, storage_t> p_in;

    auto kcache_stencil = gridtools::make_computation<backend_t>(m_grid,
        p_in{} = m_in,
        gridtools::make_multistage(execute<forward>(),
            define_caches(cache<K, cache_io_policy::fill_and_flush, kfull>(p_in())),
            gridtools::make_stage<do_nothing>(p_in())));

    kcache_stencil.run();

#if FLOAT_PRECISION == 4
    verifier verif(1e-6);
#else
    verifier verif(1e-10);
#endif
    array<array<uint_t, 2>, 3> halos{{{0, 0}, {0, 0}, {0, 0}}};

    m_in.sync();
    ASSERT_TRUE(verif.verify(m_grid, m_ref, m_in, halos));
}
