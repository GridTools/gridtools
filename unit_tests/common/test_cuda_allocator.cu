#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <gridtools/common/cuda_allocator.hpp>
#include <gridtools/tools/backend_select.hpp>

#include "../test_helper.hpp"
#include <gtest/gtest.h>

namespace gridtools {
    namespace {
        __global__ void test_allocated(float_type *data) { *data = 1; }

        TEST(simple_cuda_allocator, test) {
            // TODO use test functionality
            cuda_allocator alloc;
            auto ptr_holder = alloc.allocate<float_type>(1);

            float_type *ptr = ptr_holder();
            float_type data;

            test_allocated<<<1, 1>>>(ptr);
            hipMemcpy(&data, ptr, sizeof(float_type), hipMemcpyDeviceToHost);
            ASSERT_EQ(1, data);
        }
    } // namespace
} // namespace gridtools
