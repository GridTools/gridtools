#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "test_cxx11_tuple.hpp"

#ifdef CXX11_ENABLED

__global__
void test_tuple_kernel(bool* result)
{
    test_tuple_elements(result);
}

TEST(tuple, test_elements) {
    bool result;
    bool* resultDevice;
    hipMalloc(&resultDevice, sizeof(bool));

    // clang-format off
    test_tuple_kernel<<<1,1>>>(resultDevice);
    // clang-format on

    hipMemcpy(&result, resultDevice, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_TRUE(result);
}

#endif
