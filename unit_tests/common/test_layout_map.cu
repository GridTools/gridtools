#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "test_layout_map.hpp"

__global__
void test_layout_accessors_kernel(bool* result)
{
    test_layout_accessors(result);
}

__global__
void test_layout_findval_kernel(bool* result)
{
    test_layout_find_val(result);
}

TEST(layout_map_cuda, test_layout_accessors) {
    bool result;
    bool* resultDevice;
    hipMalloc(&resultDevice, sizeof(bool));

    // clang-format off
    test_layout_accessors_kernel<<<1,1>>>(resultDevice);
    // clang-format on

    hipMemcpy(&result, resultDevice, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_TRUE(result);
}

TEST(layout_map_cuda, test_layout_findval) {
    bool result;
    bool* resultDevice;
    hipMalloc(&resultDevice, sizeof(bool));

    // clang-format off
    test_layout_findval_kernel<<<1,1>>>(resultDevice);
    // clang-format on

    hipMemcpy(&result, resultDevice, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_TRUE(result);
}
