#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include "../cuda_test_helper.hpp"
#include <gridtools/stencil_composition/backend_cuda/shared_allocator.hpp>

#include <gtest/gtest.h>

namespace {
    template <typename PtrHolder>
    __device__ uint64_t get_ptr(PtrHolder ptr_holder) {
        return reinterpret_cast<uint64_t>(ptr_holder());
    }

    TEST(shared_allocator, alignment) {
        gridtools::shared_allocator allocator;
        EXPECT_EQ(0, allocator.size());

        using alloc1_t = char[14];
        auto alloc1 = allocator.allocate<alloc1_t>(7);

        using alloc2_t = double;
        auto alloc2 = allocator.allocate<alloc2_t>(4);

        using alloc3_t = double;
        auto alloc3 = allocator.allocate<alloc3_t>(1);

        auto ptr1 = gridtools::on_device::exec_with_shared_memory(
            allocator.size(), MAKE_CONSTANT(get_ptr<decltype(alloc1)>), alloc1);
        auto ptr2 = gridtools::on_device::exec_with_shared_memory(
            allocator.size(), MAKE_CONSTANT(get_ptr<decltype(alloc2)>), alloc2);
        auto ptr3 = gridtools::on_device::exec_with_shared_memory(
            allocator.size(), MAKE_CONSTANT(get_ptr<decltype(alloc3)>), alloc3);

        // check alignment for all allocations
        EXPECT_EQ(ptr1 % alignof(alloc1_t), 0);
        EXPECT_EQ(ptr2 % alignof(alloc2_t), 0);
        EXPECT_EQ(ptr3 % alignof(alloc3_t), 0);
    }

    template <class T>
    ptrdiff_t get_offset(gridtools::shared_allocator const &allocator, T const &alloc1, T const &alloc2) {
        auto offset1 = gridtools::on_device::exec_with_shared_memory(
            allocator.size(), MAKE_CONSTANT(get_ptr<T>), alloc1);
        auto offset2 = gridtools::on_device::exec_with_shared_memory(
            allocator.size(), MAKE_CONSTANT(get_ptr<T>), alloc2);
        return offset2 - offset1;
    }

    TEST(shared_allocator, pointer_arithmetics) {
        gridtools::shared_allocator allocator;
        auto some_alloc = allocator.allocate<double>(32);
        auto another_alloc = allocator.allocate<double>(32);

        EXPECT_EQ(get_offset(allocator, another_alloc, another_alloc + 3), 3 * (int)sizeof(double));
    }

    template <class PtrHolderFloat, class PtrHolderDouble>
    __device__ int fill_and_check_test(PtrHolderFloat alloc1, PtrHolderDouble alloc2, bool *result) {
        static_assert(std::is_same<decltype(alloc1()), float *>::value, "");
        static_assert(std::is_same<decltype(alloc2()), double *>::value, "");

        auto ptr1 = alloc1();
        auto ptr2 = alloc2();

        ptr1[threadIdx.x] = 100 * blockIdx.x + threadIdx.x;
        ptr2[threadIdx.x] = 10000 + 100 * blockIdx.x + threadIdx.x;
        __syncthreads();

        if (threadIdx.x == 0) {
            bool local_result = true;
            for (int i = 0; i < 32; ++i)
                local_result &= (ptr1[i] == 100 * blockIdx.x + i && ptr2[i] == 10000 + 100 * blockIdx.x + i);

            result[blockIdx.x] = local_result;
        }
        return 0;
    }

    TEST(shared_allocator, fill_and_check) {
        gridtools::shared_allocator allocator;
        auto float_allocation = allocator.allocate<float>(32);
        auto double_allocation = allocator.allocate<double>(32);

        bool *result;
        hipMallocManaged(&result, 2 * sizeof(bool));

        gridtools::on_device::exec_with_shared_memory<2, 32>(allocator.size(),
            MAKE_CONSTANT((fill_and_check_test<decltype(float_allocation), decltype(double_allocation)>)),
            float_allocation,
            double_allocation,
            result);

        EXPECT_TRUE(result[0]);
        EXPECT_TRUE(result[1]);

        hipFree(result);
    }

} // namespace
