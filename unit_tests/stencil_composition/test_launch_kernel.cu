#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <gridtools/stencil_composition/backend_cuda/launch_kernel.hpp>

#include <gtest/gtest.h>

#include <gridtools/common/cuda_util.hpp>
#include <gridtools/common/defs.hpp>
#include <gridtools/common/host_device.hpp>
#include <gridtools/meta.hpp>
#include <gridtools/stencil_composition/extent.hpp>

namespace gridtools {
    namespace cuda {
        template <class Extent, int_t IBlockSize, int_t JBlockSize>
        struct kernel_f {
            int *m_failures;
            int_t m_i_size;
            int_t m_j_size;

            template <class Validator>
            GT_FUNCTION_DEVICE void operator()(int_t iblock, int_t jblock, Validator is_valid) const {
                int_t i_block_size =
                    (blockIdx.x + 1) * IBlockSize < m_i_size ? IBlockSize : m_i_size - blockIdx.x * IBlockSize;
                int_t j_block_size =
                    (blockIdx.y + 1) * JBlockSize < m_j_size ? JBlockSize : m_j_size - blockIdx.y * JBlockSize;
                bool expected = Extent::iminus::value <= iblock && Extent::iplus::value + i_block_size > iblock &&
                                Extent::jminus::value <= jblock && Extent::jplus::value + j_block_size > jblock;
                bool actual = is_valid(Extent());
                if (actual == expected)
                    return;
                atomicAdd(m_failures, 1);
                printf("failure at {%d,%d} of block {%d,%d}: false %s\n",
                    iblock,
                    jblock,
                    blockIdx.x,
                    blockIdx.y,
                    actual ? "positive" : "negative");
            }
        };

        template <class MaxExtent, class Extent, int_t IBlockSize, int_t JBlockSize>
        void do_test(int_t i_size, int_t j_size) {
            auto failures = cuda_util::make_clone(0);
            kernel_f<Extent, IBlockSize, JBlockSize> kernel = {failures.get(), i_size, j_size};
            launch_kernel<MaxExtent, IBlockSize, JBlockSize>(i_size, j_size, 1, kernel, 0);
            EXPECT_EQ(0, cuda_util::from_clone(failures));
        }

        TEST(launch_kernel, simplest) { do_test<extent<>, extent<>, 32, 8>(128, 128); }

        TEST(launch_kernel, rounded_sizes) { do_test<extent<-2, 2, -1, 3>, extent<-1, 1, 0, 2>, 32, 8>(128, 128); }

        TEST(launch_kernel, hori_diff) { do_test<extent<-1, 1, -1, 1>, extent<-1, 1, -1, 1>, 32, 8>(128, 128); }

        TEST(launch_kernel, hori_diff_small_size) { do_test<extent<-1, 1, -1, 1>, extent<-1, 1, -1, 1>, 32, 8>(5, 5); }

        TEST(launch_kernel, max_extent) { do_test<extent<-2, 2, -1, 3>, extent<-2, 2, -1, 3>, 32, 8>(123, 50); }

        TEST(launch_kernel, zero_extent) { do_test<extent<-2, 2, -1, 3>, extent<>, 32, 8>(123, 50); }

        TEST(launch_kernel, reduced_extent) { do_test<extent<-2, 2, -1, 3>, extent<-1, 1, 0, 2>, 32, 8>(123, 50); }
    } // namespace cuda
} // namespace gridtools
