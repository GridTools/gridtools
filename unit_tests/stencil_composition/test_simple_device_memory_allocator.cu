#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <gridtools/stencil_composition/backend_cuda/simple_device_memory_allocator.hpp>

#include "../cuda_test_helper.hpp"
#include <gridtools/common/integral_constant.hpp>
#include <gridtools/tools/backend_select.hpp>

#include <gtest/gtest.h>

namespace gridtools {
    namespace {

        template <typename PtrHolder>
        __device__ bool check_allocation(PtrHolder ptr_holder) {
            auto &ref = *ptr_holder();
            ref = 1.;
            return ref == 1.;
        }

        template <typename PtrHolder>
        __global__ void test_allocated(PtrHolder testee, bool *result) {}

        TEST(simple_device_memory_allocator, test) {
            simple_device_memory_allocator alloc;
            auto ptr_holder = alloc.allocate<float_type>(1);

            auto result = gridtools::on_device::exec(
                GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&check_allocation<decltype(ptr_holder)>), ptr_holder);
            ASSERT_TRUE(result);
        }
    } // namespace
} // namespace gridtools
