#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2019, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <gridtools/stencil_composition/backend_cuda/simple_device_memory_allocator.hpp>
#include <gridtools/tools/backend_select.hpp>

#include "../test_helper.hpp"
#include <gtest/gtest.h>

namespace gridtools {
    namespace {
        template <typename PtrHolder>
        __global__ void test_allocated(PtrHolder data) {
            *(data()) = 1;
        }

        TEST(simple_device_memory_allocator, test) {
            simple_device_memory_allocator alloc;
            auto ptr_holder = alloc.allocate<float_type>(1);

            test_allocated<<<1, 1>>>(ptr_holder);
            float_type data;
            hipMemcpy(&data, alloc.ptrs()[0].get(), sizeof(float_type), hipMemcpyDeviceToHost);
            ASSERT_EQ(1, data);
        }
    } // namespace
} // namespace gridtools
