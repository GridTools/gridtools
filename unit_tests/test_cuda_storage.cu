#include "hip/hip_runtime.h"
/*
 * File:   test_domain.cpp
 * Author: mbianco
 *
 * Created on February 14, 2014, 4:18 PM
 *
 * Test cuda_storage features
 */

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <common/gpu_clone.hpp>
#include <storage/hybrid_pointer.hpp>
#include <stencil-composition/backend.hpp>
#include <common/layout_map.hpp>
#include <common/defs.hpp>
#include <stencil-composition/backend.hpp>

using gridtools::uint_t;
using gridtools::int_t;

#ifdef __HIPCC__
template <typename T>
__global__
void add_on_gpu(T * ptr, uint_t d1, uint_t d2, uint_t d3) {
    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
                (*ptr)(i,j,k) = -i-j-k;
            }
        }
    }
}
#endif

using namespace gridtools;
using namespace enumtype;
bool test_cuda_storage() {

    typedef gridtools::backend<gridtools::enumtype::Cuda, gridtools::enumtype::Naive > backend_t;
    typedef gridtools::backend<Cuda, Block>::storage_type<float_type, gridtools::layout_map<0,1,2> > ::type storage_type;

    uint_t d1 = 3;
    uint_t d2 = 3;
    uint_t d3 = 3;

    storage_type data(d1,d2,d3,-1., "data");

    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
                data(i,j,k) = i+j+k;
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    data.h2d_update();
    data.clone_to_gpu();
#ifdef __HIPCC__
    add_on_gpu<<<1,1>>>(data.gpu_object_ptr, d1, d2, d3);
    hipDeviceSynchronize();
#endif
    data.d2h_update();

    bool same = true;
    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
                if (data(i,j,k) != -i-j-k)
                    same = false;
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    return same;
}
