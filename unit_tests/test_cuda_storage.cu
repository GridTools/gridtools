#include "hip/hip_runtime.h"
/*
 * File:   test_domain.cpp
 * Author: mbianco
 *
 * Created on February 14, 2014, 4:18 PM
 *
 * Test cuda_storage features
 */

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stencil-composition/stencil-composition.hpp>

using gridtools::uint_t;
using gridtools::int_t;

template <typename T, typename U>
__global__
void add_on_gpu(U* meta, T * ptr, uint_t d1, uint_t d2, uint_t d3) {
    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
                ptr->fields_view()[0][meta->index(i,j,k)] = -i-j-k;
            }
        }
    }
}

using namespace gridtools;
using namespace enumtype;
bool test_cuda_storage() {

    typedef backend<Cuda, Block > backend_t;
    typedef backend_t::storage_type<float_type, backend_t::storage_info<0,layout_map<0,1,2> > > ::type storage_type;

    uint_t d1 = 3;
    uint_t d2 = 3;
    uint_t d3 = 3;

    typename storage_type::storage_info_type meta_(d1,d2,d3);
    storage_type data(meta_, -1., "data"); //allocate on GPU

    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
                data(i,j,k) = i+j+k;
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    data.h2d_update(); //copy to GPU
    data.clone_to_device();
    meta_.clone_to_device();//copy meta information to the GPU

    add_on_gpu<<<1,1>>>(meta_.gpu_object_ptr, data.gpu_object_ptr, d1, d2, d3);
    hipDeviceSynchronize();

    data.d2h_update();

    bool same = true;
    for (uint_t i = 0; i < d1; ++i) {
        for (uint_t j = 0; j < d2; ++j) {
            for (uint_t k = 0; k < d3; ++k) {
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
                if (data(i,j,k) != -i-j-k)
                    same = false;
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    return same;
}
