/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include <gridtools/fn/backend2/gpu.hpp>

#include <gtest/gtest.h>

#include <gridtools/fn/scan.hpp>
#include <gridtools/sid/composite.hpp>
#include <gridtools/sid/synthetic.hpp>

namespace gridtools::fn::backend {
    namespace {
        using namespace literals;
        using sid::property;

        template <int I>
        using int_t = integral_constant<int, I>;

        struct sum_scan : fwd {
            static GT_FUNCTION constexpr auto body() {
                return scan_pass(
                    [](auto acc, auto const &iter) { return tuple(get<0>(acc) + *iter, get<1>(acc) * *iter); },
                    [](auto acc) { return get<0>(acc); });
            }
        };

        struct make_iterator_mock {
            GT_FUNCTION auto operator()() const {
                return [](auto tag, auto const &ptr, auto const &strides) { return at_key<decltype(tag)>(ptr); };
            }
        };

        TEST(backend_gpu, apply_column_stage) {
            auto in = cuda_util::cuda_malloc<int>(5 * 7 * 3);
            auto out = cuda_util::cuda_malloc<int>(5 * 7 * 3);
            int inh[5][7][3], outh[5][7][3] = {};
            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 7; ++j)
                    for (int k = 0; k < 3; ++k)
                        inh[i][j][k] = 21 * i + 3 * j + k;
            hipMemcpy(in.get(), inh, 5 * 7 * 3 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::make_simple_ptr_holder(x))
                    .set<property::strides>(tuple(21_c, 3_c, 1_c));
            };

            auto composite =
                sid::composite::keys<int_t<0>, int_t<1>>::make_values(as_synthetic(out.get()), as_synthetic(in.get()));

            auto sizes = hymap::keys<int_t<0>, int_t<1>, int_t<2>>::values<int_t<5>, int_t<7>, int_t<3>>();

            column_stage<int_t<1>, sum_scan, make_iterator_mock, 0, 1> cs;

            using block_sizes_t = meta::list<meta::list<int_t<0>, int_t<4>>, meta::list<int_t<2>, int_t<2>>>;

            apply_column_stage<int_t<1>>(gpu<block_sizes_t>(), sizes, cs, composite, tuple(42, 1));

            hipMemcpy(outh, out.get(), 5 * 7 * 3 * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < 5; ++i)
                for (int k = 0; k < 3; ++k) {
                    int res = 42;
                    for (int j = 0; j < 7; ++j) {
                        res += inh[i][j][k];
                        EXPECT_EQ(outh[i][j][k], res);
                    }
                }
        }
    } // namespace
} // namespace gridtools::fn::backend
